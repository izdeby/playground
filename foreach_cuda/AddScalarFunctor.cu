#include "hip/hip_runtime.h"
#include <torch/hip/hip_runtime.h>
#include "Utils.cuh"

template<typename x_t, typename out_t>
struct AddScalarFunctor
{
    __device__ void operator() (
        int chunk_size,
        TensorListMetadata<2>& tl,
        float scale) 
        {
            int tensor_loc = tl.block_to_tensor[blockIdx.x];
            int chunk_idx = tl.block_to_chunk[blockIdx.x];
            int n = tl.sizes[tensor_loc];
            //printf("[AxpbyFunctor] \n\tblockIdx.x = %i\n\ttensor_loc = %i\n\tchunk_idx = %i\n", blockIdx.x, tensor_loc, chunk_idx);
            
            x_t* x = (x_t*)tl.addresses[0][tensor_loc];
            x += chunk_idx*chunk_size;

            out_t* out = (out_t*)tl.addresses[1][tensor_loc];
            out += chunk_idx*chunk_size;

            n -= chunk_idx*chunk_size;

            x_t r_x[ILP];
            out_t r_out[ILP];

            // to make things simple, we put aligned case in a different code path
            if(n % ILP == 0 && chunk_size % ILP == 0 && is_aligned(x) && is_aligned(out))
            {
                //printf("[AxpbyFunctor] case 1\n");
                for(int i_start = threadIdx.x; i_start*ILP < n && i_start*ILP < chunk_size; i_start += blockDim.x)
                {
                    // load
                    load_store(r_x, x, 0 , i_start);
#pragma unroll
                    for(int ii = 0; ii < ILP; ii++)
                    {
                        r_out[ii] = static_cast<float>(r_x[ii]) + scale;
                    }
                    // store
                    load_store(out, r_out, i_start , 0);
                }
            }
            else
            {
                //printf("[AxpbyFunctor] case 2\n");
                // Non-divergent exit condition for __syncthreads, not necessary here
                for(int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x*ILP)
                {
#pragma unroll
                    for(int ii = 0; ii < ILP; ii++)
                    {
                        r_x[ii] = 0;
                        int i = i_start + threadIdx.x + ii * blockDim.x;
                        if(i < n && i < chunk_size)
                        {
                            r_x[ii] = x[i];
                        }
                    }
#pragma unroll
                    for(int ii = 0; ii < ILP; ii++)
                    {
                        r_out[ii] = static_cast<float>(r_x[ii]) + scale;
                    }
#pragma unroll
                    for(int ii = 0; ii < ILP; ii++)
                    {
                    int i = i_start + threadIdx.x + ii*blockDim.x;
                    if(i < n && i < chunk_size)
                        out[i] = r_out[ii];
                    }
                }
            }
        }
};