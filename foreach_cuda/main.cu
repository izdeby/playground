
#include <iostream>
#include <math.h>
#include <torch/torch.h>
#include <torch/hip/hip_runtime.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int main(void)
{
  //std::cout << torch.cuda.is_available() << std::endl;
  torch::Device device(torch::kCUDA);
  torch::Tensor tensor = torch::eye(3, device);
  std::cout << tensor << std::endl;
  return 0;
}